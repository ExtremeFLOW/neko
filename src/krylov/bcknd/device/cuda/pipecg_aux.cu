#include "pipecg_kernel.h"
#include <device/device_config.h>
#include <device/cuda/check.h>

/**
 * @todo Make sure that this gets deleted at some point...
 */
real *buf1 = NULL;
real *buf2 = NULL;
real *buf3 = NULL;
real *buf_d1 = NULL;
real *buf_d2 = NULL;
real *buf_d3 = NULL;

extern "C" {
  
  void cuda_cg_update_xp(void *x, void *p, void *u, void *alpha, void *beta,
			 int *p_cur, int *p_space, int *n) {
	
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);
    const int nb = ((*n) + 1024 - 1)/ 1024;
    
    cg_update_xp_kernel<real>
      <<<nblcks, nthrds>>>((real *) x, (real *) p,(real **) u, (real *) alpha,
			   (real *) beta, *p_cur, *p_space, *n);
    CUDA_CHECK(hipGetLastError());

  }

  void cuda_pipecg_vecops(void *p, void *q, void *r, void *s, void *u1,
                            void *u2, void *w, void *z, 
                            void *ni, void *mi, real *alpha, 
                            real *beta, void *mult, 
                            real *reduction, int *n) {
	
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);
    const int nb = ((*n) + 1024 - 1)/ 1024;
    
    if (!buf1){
      buf1 = (real *) malloc(nb * sizeof(real));
      buf2 = (real *) malloc(nb * sizeof(real));
      buf3 = (real *) malloc(nb * sizeof(real));
      CUDA_CHECK(hipMalloc(&buf_d1, nb*sizeof(real)));
      CUDA_CHECK(hipMalloc(&buf_d2, nb*sizeof(real)));
      CUDA_CHECK(hipMalloc(&buf_d3, nb*sizeof(real)));
    }
     
    pipecg_vecops_kernel<real>
      <<<nblcks, nthrds>>>((real *) p, (real *) q,
			   (real *) r, (real *) s,
			   (real *) u1, (real *) u2,
			   (real *) w, (real *) z,
			   (real *) ni, (real *) mi, 
			   *alpha, *beta, (real *)mult, 
			   buf_d1, buf_d2, buf_d3, *n);
    
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipMemcpy(buf1, buf_d1, nb * sizeof(real),
			  hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(buf2, buf_d2, nb * sizeof(real),
			  hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(buf3, buf_d3, nb * sizeof(real),
			  hipMemcpyDeviceToHost));

    real res1 = 0.0;
    real res2 = 0.0;
    real res3 = 0.0;
    for (int i = 0; i < nb; i++) {
      res1 += buf1[i];
      res2 += buf2[i];
      res3 += buf3[i];
    }

    reduction[0] = res1;
    reduction[1] = res2;
    reduction[2] = res3;
  }
}
