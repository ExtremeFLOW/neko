
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

template<const int LX>
__global__ void jacobi_kernel(double * __restrict__ du,
			      const double * __restrict__ dxt,
			      const double * __restrict__ dyt,
			      const double * __restrict__ dzt,
			      const double * __restrict__ G11,
			      const double * __restrict__ G22,
			      const double * __restrict__ G33,
			      const double * __restrict__ G12,
			      const double * __restrict__ G13,
			      const double * __restrict__ G23,
			      const int nel) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  const int e = idx / (LX*LX*LX);
  const int k = idx / (LX*LX) % LX;
  const int j = idx / LX % LX;
  const int i = idx % LX;

  if (e >= nel)
    return;

  double d = 0.0;

  for (int l = 0; l < LX; l++) {
    double g = G11[l + LX*j + LX*LX*k + LX*LX*LX*e];
    double t = dxt[i + LX*l];
    d += g*t*t;
  }

  for (int l = 0; l < LX; l++) {
    double g = G22[i + LX*l + LX*LX*k + LX*LX*LX*e];
    double t = dyt[j + LX*l];
    d += g*t*t;
  }

  for (int l = 0; l < LX; l++) {
    double g = G33[i + LX*j + LX*LX*l + LX*LX*LX*e];
    double t = dzt[k + LX*l];
    d += g*t*t;
  }

  // Corrections for deformed elements
  if (i == 0 || i == LX-1) {
    d += G12[i + LX*j + LX*LX*k + LX*LX*LX*e] * dxt[i + LX*i] * dyt[j + LX*j];
    d += G13[i + LX*j + LX*LX*k + LX*LX*LX*e] * dxt[i + LX*i] * dzt[k + LX*k];
  }

  if (j == 0 || j == LX-1) {
    d += G12[i + LX*j + LX*LX*k + LX*LX*LX*e] * dyt[i + LX*i] * dxt[j + LX*j];
    d += G23[i + LX*j + LX*LX*k + LX*LX*LX*e] * dyt[i + LX*i] * dzt[k + LX*k];
  }

  if (k == 0 || k == LX-1) {
    d += G13[i + LX*j + LX*LX*k + LX*LX*LX*e] * dzt[i + LX*i] * dxt[j + LX*j];
    d += G23[i + LX*j + LX*LX*k + LX*LX*LX*e] * dzt[i + LX*i] * dyt[k + LX*k];
  }

  du[idx] = d;
}

extern "C" {
  void cuda_jacobi_update(void *d,
			  void *dxt, void *dyt, void *dzt,
			  void *G11, void *G22, void *G33,
			  void *G12, void *G13, void *G23,
			  int *nel, int *lxp) {

    const int lx = *lxp;
    const int threads = 1024;
    const int blocks = ((*nel * lx*lx*lx) + threads - 1) / threads;

#define CASE(N)\
    case N:\
    jacobi_kernel<N><<<blocks, threads>>>(\
	(double*)d,\
	(double*)dxt, (double*)dyt, (double*)dzt,\
	(double*)G11, (double*)G22, (double*)G33,\
	(double*)G12, (double*)G13, (double*)G23,\
	*nel);\
    break

    switch (lx) {
    CASE(1);
    CASE(2);
    CASE(3);
    CASE(4);
    CASE(5);
    CASE(6);
    CASE(7);
    CASE(8);
    CASE(9);
    CASE(10);
    CASE(11);
    CASE(12);
    CASE(13);
    CASE(14);
    CASE(15);
    default:
      fprintf(stderr, __FILE__ ": size not supported: %d\n", lx);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, __FILE__ ": %s\n", hipGetErrorString(err));
    }
  }
}
