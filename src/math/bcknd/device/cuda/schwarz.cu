/*
 Copyright (c) 2022, The Neko Authors
 All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions
 are met:

   * Redistributions of source code must retain the above copyright
     notice, this list of conditions and the following disclaimer.

   * Redistributions in binary form must reproduce the above
     copyright notice, this list of conditions and the following
     disclaimer in the documentation and/or other materials provided
     with the distribution.

   * Neither the name of the authors nor the names of its
     contributors may be used to endorse or promote products derived
     from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 POSSIBILITY OF SUCH DAMAGE.
*/

#include "schwarz_kernel.h"
#include <device/device_config.h>
#include <device/cuda/check.h>
#include <stdio.h>

extern "C" {

  /** 
   * Fortran wrapper for device extrude
   */
  void cuda_schwarz_extrude(void *arr1, int * l1, real * f1,
                            void *arr2, int * l2, real * f2,
                            int * nx, int * nel) {
    
    const dim3 nthrds((*nx-2)*(*nx-2), 1, 1);
    const dim3 nblcks((*nel), 1, 1);

#define CASE(NX)                                     \
    case NX:                                         \
    schwarz_extrude_kernel<real,NX>                  \
    <<<nblcks, nthrds>>>((real *) arr1,* l1, * f1,   \
                         (real *) arr2, *l2, *f2 );  \
    CUDA_CHECK(hipGetLastError());                  \
    break;

    switch(*nx) {
      CASE(3);
      CASE(4);
      CASE(5);
      CASE(6);
      CASE(7);
      CASE(8);
      CASE(9);
      CASE(10);
      CASE(11);
      CASE(12);
      CASE(13);
      CASE(14);
    default:
      {
        fprintf(stderr, __FILE__ ": size not supported: %d\n", *nx);
        exit(1);
      }
   }

  } 

  void cuda_schwarz_toext3d(void *a, void *b,int * nx, int * nel){
    
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks((*nel), 1, 1);

    schwarz_toext3d_kernel<real>
    <<<nblcks, nthrds>>>((real *) a,(real *) b, * nx);  
    CUDA_CHECK(hipGetLastError());
  } 

  void cuda_schwarz_toreg3d(void *b, void *a,int * nx, int * nel){
    
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks((*nel), 1, 1);

    schwarz_toreg3d_kernel<real>
    <<<nblcks, nthrds>>>((real *) b,(real *) a, * nx);  
    CUDA_CHECK(hipGetLastError());
  } 

}
