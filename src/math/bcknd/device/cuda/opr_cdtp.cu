/*
 Copyright (c) 2021-2023, The Neko Authors
 All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions
 are met:

   * Redistributions of source code must retain the above copyright
     notice, this list of conditions and the following disclaimer.

   * Redistributions in binary form must reproduce the above
     copyright notice, this list of conditions and the following
     disclaimer in the documentation and/or other materials provided
     with the distribution.

   * Neither the name of the authors nor the names of its
     contributors may be used to endorse or promote products derived
     from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 POSSIBILITY OF SUCH DAMAGE.
*/

#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include "cdtp_kernel.h"
#include <device/device_config.h>
#include <device/cuda/check.h>

extern "C" {
  #include <common/neko_log.h>
}

template < const int >
int tune_cdtp(void *dtx, void *x,
              void *dr, void *ds, void *dt,
              void *dxt, void *dyt, void *dzt,
              void *B, void *jac, int *nel, int *lx);

extern "C" {

  /** 
   * Fortran wrapper for device cuda \f$ D^T X \f$
   */
  void cuda_cdtp(void *dtx, void *x,
                 void *dr, void *ds, void *dt,
                 void *dxt, void *dyt, void *dzt,
                 void *B, void *jac, int *nel, int *lx) {
    
    static int autotune[17] = { 0 };
    
    const dim3 nthrds_1d(1024, 1, 1);
    const dim3 nthrds_kstep((*lx), (*lx), 1);
    const dim3 nblcks((*nel), 1, 1);

#define CASE_1D(LX)                                                             \
    cdtp_kernel_1d<real, LX, 1024>                                              \
      <<<nblcks, nthrds_1d>>>((real *) dtx, (real *) x,                         \
                              (real *) dr, (real *) ds, (real *) dt,            \
                              (real *) dxt, (real *) dyt, (real *) dzt,         \
                              (real *) B, (real *) jac);                        \
    CUDA_CHECK(hipGetLastError());

#define CASE_KSTEP(LX)                                                          \
    cdtp_kernel_kstep<real, LX>                                                 \
      <<<nblcks, nthrds_kstep>>>((real *) dtx, (real *) x,                      \
                                 (real *) dr, (real *) ds, (real *) dt,         \
                                 (real *) dxt, (real *) dyt, (real *) dzt,      \
                                 (real *) B, (real *) jac);                     \
    CUDA_CHECK(hipGetLastError());

#define CASE(LX)                                                                \
    case LX:                                                                    \
      if(autotune[LX] == 0 ) {                                                  \
        autotune[LX]=tune_cdtp<LX>(dtx, x,                                      \
                                   dr, ds, dt,                                  \
                                   dxt, dyt, dzt,                               \
                                   B, jac, nel, lx);                            \
      } else if (autotune[LX] == 1 ) {                                          \
        CASE_1D(LX);                                                            \
      } else if (autotune[LX] == 2 ) {                                          \
        CASE_KSTEP(LX);                                                         \
      }                                                                         \
      break

#define CASE_LARGE(LX)                                                          \
    case LX:                                                                    \
      CASE_KSTEP(LX);                                                           \
      break


    if ((*lx) < 13) {      
      switch(*lx) {
        CASE(2);
        CASE(3);
        CASE(4);
        CASE(5);
        CASE(6);
        CASE(7);
        CASE(8);
        CASE(9);
        CASE(10);
        CASE(11);
        CASE(12);
      default:
        {
          fprintf(stderr, __FILE__ ": size not supported: %d\n", *lx);
          exit(1);
        }
      }
    }
    else {
      switch(*lx) {
        CASE_LARGE(13);
        CASE_LARGE(14);
        CASE_LARGE(15);
        CASE_LARGE(16);
      default:
        {
          fprintf(stderr, __FILE__ ": size not supported: %d\n", *lx);
          exit(1);
        }
      } 
    }
  } 
}    

template < const int LX >
int tune_cdtp(void *dtx, void *x,
              void *dr, void *ds, void *dt,
              void *dxt, void *dyt, void *dzt,
              void *B, void *jac, int *nel, int *lx) {
  hipEvent_t start,stop;
  float time1,time2;
  int retval;

  const dim3 nthrds_1d(1024, 1, 1);
  const dim3 nthrds_kstep((*lx), (*lx), 1);
  const dim3 nblcks((*nel), 1, 1);
  
  char *env_value = NULL;
  char neko_log_buf[80];
  
  env_value=getenv("NEKO_AUTOTUNE");

  sprintf(neko_log_buf, "Autotune cdtp (lx: %d)", *lx);
  log_section(neko_log_buf);
  
  if(env_value) {
    if( !strcmp(env_value,"1D") ) {
      CASE_1D(LX);       
      sprintf(neko_log_buf,"Set by env : 1 (1D)");
      log_message(neko_log_buf);
      log_end_section();
      return 1;
    } else if( !strcmp(env_value,"KSTEP") ) {
      CASE_KSTEP(LX);
      sprintf(neko_log_buf,"Set by env : 2 (KSTEP)");
      log_message(neko_log_buf);
      log_end_section();
      return 2;
    } else {       
       sprintf(neko_log_buf, "Invalid value set for NEKO_AUTOTUNE");
       log_error(neko_log_buf);
    }
  }

  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  hipEventRecord(start,0);
   
  for(int i = 0; i < 100; i++) {
    CASE_1D(LX);
  }
  
  hipEventRecord(stop,0); 
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time1, start, stop);
  
  hipEventRecord(start,0);
   
  for(int i = 0; i < 100; i++) {
     CASE_KSTEP(LX);
   }
  
  hipEventRecord(stop,0); 
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time2, start, stop);
  
  if(time1 < time2) {
     retval = 1;
  } else {
    retval = 2;
  }

  sprintf(neko_log_buf, "Chose      : %d (%s)", retval,
          (retval > 1 ? "KSTEP" : "1D"));
  log_message(neko_log_buf);
  log_end_section();
  return retval;
}
