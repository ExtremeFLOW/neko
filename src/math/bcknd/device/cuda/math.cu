#include "math_kernel.h"
#include <device/device_config.h>

extern "C" {

  /** Fortran wrapper for copy
   * Copy a vector \f$ a = b \f$
   */
  void cuda_copy(void *a, void *b, int *n) {
    hipMemcpyAsync(a, b, (*n) * sizeof(real), hipMemcpyDeviceToDevice);
  }

  /** Fortran wrapper for rzero
   * Zero a real vector
   */
  void cuda_rzero(void *a, int *n) {
    hipMemsetAsync(a, 0, (*n) * sizeof(real));
  }

  /** Fortran wrapper for rone
   * Set all elements to one
   */
  void cuda_rone(void *a, int *n) {
    hipMemsetAsync(a, 1, (*n) * sizeof(real));
  }

  /** Fortran wrapper for cmult
   * Multiplication by constant c \f$ a = c \cdot a \f$
   */
  void cuda_cmult(void *a, real *c, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    cmult_kernel<real><<<nblcks, nthrds>>>((real *) a,
					   *c, *n);

  }

  /** Fortran wrapper for cfill
   * Set all elements to a constant c \f$ a = c \f$
   */
  void cuda_cfill(void *a, real *c, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    cfill_kernel<real><<<nblcks, nthrds>>>((real *) a,
					   *c, *n);

  }
  
  /**
   * Fortran wrapper for add2s1
   * Vector addition with scalar multiplication \f$ a = c_1 a + b \f$
   * (multiplication on first argument) 
   */
  void cuda_add2s1(void *a, void *b, real *c1, int *n) {
    
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    add2s1_kernel<real><<<nblcks, nthrds>>>((real *) a,
					    (real *) b,
					    *c1, *n);
    
  }

  /**
   * Fortran wrapper for add2s2
   * Vector addition with scalar multiplication \f$ a = a + c_1 b \f$
   * (multiplication on second argument) 
   */
  void cuda_add2s2(void *a, void *b, real *c1, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    add2s2_kernel<real><<<nblcks, nthrds>>>((real *) a,
					    (real *) b,
					    *c1, *n);

  }

  /**
   * Fortran wrapper for add3s2
   * Vector addition with scalar multiplication \f$ a = c_1 b + c_2 c \f$
   * (multiplication on second argument) 
   */
  void cuda_add3s2(void *a, void *b, void *c, real *c1, real *c2, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    add3s2_kernel<real><<<nblcks, nthrds>>>((real *) a,
					    (real *) b,
					    (real *) c,
					    *c1, *c2, *n);

  }

 
  /**
   * Fortran wrapper for invcol1
   * Invert a vector \f$ a = 1 / a \f$
   */
  void cuda_invcol1(void *a, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    invcol1_kernel<real><<<nblcks, nthrds>>>((real *) a,
					     *n);
  }
  /**
   * Fortran wrapper for invcol2
   * Vector division \f$ a = a / b \f$
   */
  void cuda_invcol2(void *a, void *b, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    invcol2_kernel<real><<<nblcks, nthrds>>>((real *) a,
					       (real *) b, *n);
  }
  
  /**
   * Fortran wrapper for col2
   * Vector multiplication with 2 vectors \f$ a = a \cdot b \f$
   */
  void cuda_col2(void *a, void *b, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    col2_kernel<real><<<nblcks, nthrds>>>((real *) a, 
					    (real *) b, *n);
  }
  
  /**
   * Fortran wrapper for col3
   * Vector multiplication with 3 vectors \f$ a = b \cdot c \f$
   */
  void cuda_col3(void *a, void *b, void *c, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    col3_kernel<real><<<nblcks, nthrds>>>((real *) a, (real *) b,
					    (real *) c, *n);
  }

  /**
   * Fortran wrapper for subcol3
   * Vector multiplication with 3 vectors \f$ a = a - b \cdot c \f$
   */
  void cuda_subcol3(void *a, void *b, void *c, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    subcol3_kernel<real><<<nblcks, nthrds>>>((real *) a, (real *) b,
					     (real *) c, *n);
  }
  

  /**
   * Fortran wrapper for sub3
   * Vector subtraction \f$ a = b - c \f$
   */
  void cuda_sub3(void *a, void *b, void *c, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    sub3_kernel<real><<<nblcks, nthrds>>>((real *) a, (real *) b, 
					    (real *) c, *n);
  }

  /**
   * Fortran wrapper for addcol3
   * \f$ a = a + b * c \f$
   */
  void cuda_addcol3(void *a, void *b, void *c, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    addcol3_kernel<real><<<nblcks, nthrds>>>((real *) a, (real *) b,
					       (real *) c, *n);
  }

  /**
   * Fortran wrapper glsc3
   * Weighted inner product \f$ a^T b c \f$
   */
  real cuda_glsc3(void *a, void *b, void *c, int *n) {
	
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);
    const int nb = ((*n) + 1024 - 1)/ 1024;
    
    real * buf = (real *) malloc(nb * sizeof(real));
    real * buf_d;

    hipMalloc(&buf_d, nb*sizeof(real));
     
    glsc3_kernel<real><<<nblcks, nthrds>>>((real *) a, (real *) b,
					     (real *) c, buf_d, *n);

    hipMemcpy(buf, buf_d, nb * sizeof(real), hipMemcpyDeviceToHost);

    real res = 0.0;
    for (int i = 0; i < nb; i++) {
      res += buf[i];
    }

    free(buf);
    hipFree(buf_d);

    return res;
  }

  /**
   * Fortran wrapper glsc2
   * Weighted inner product \f$ a^T b c \f$
   */
  real cuda_glsc2(void *a, void *b, int *n) {
	
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);
    const int nb = ((*n) + 1024 - 1)/ 1024;
    
    real * buf = (real *) malloc(nb * sizeof(real));
    real * buf_d;

    hipMalloc(&buf_d, nb*sizeof(real));
     
    glsc2_kernel<real><<<nblcks, nthrds>>>((real *) a, (real *) b,
					      buf_d, *n);

    hipMemcpy(buf, buf_d, nb * sizeof(real), hipMemcpyDeviceToHost);

    real res = 0.0;
    for (int i = 0; i < nb; i++) {
      res += buf[i];
    }

    free(buf);
    hipFree(buf_d);

    return res;
  }

}
