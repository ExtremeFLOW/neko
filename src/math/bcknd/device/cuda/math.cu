#include "math_kernel.h"
#include <device/device_config.h>

extern "C" {

  /** Fortran wrapper for copy
   * Copy a vector \f$ a = b \f$
   */
  void cuda_copy(void *a, void *b, int *n) {
    hipMemcpy(a, b, (*n) * sizeof(real), hipMemcpyDeviceToDevice);
  }

  /** Fortran wrapper for rzero
   * Zero a real vector
   */
  void cuda_rzero(void *a, int *n) {
    hipMemset(a, 0, (*n) * sizeof(real));
  }

  
  /**
   * Fortran wrapper for add2s1
   * Vector addition with scalar multiplication \f$ a = c_1 a + b \f$
   * (multiplication on first argument) 
   */
  void cuda_add2s1(void *a, void *b, real *c1, int *n) {
    
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    add2s1_kernel<real><<<nblcks, nthrds>>>((real *) a,
					      (real *) b,
					      *c1, *n);
    
  }

  /**
   * Fortran wrapper for add2s2
   * Vector addition with scalar multiplication \f$ a = a + c_1 b \f$
   * (multiplication on second argument) 
   */
  void cuda_add2s2(void *a, void *b, real *c1, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    add2s2_kernel<real><<<nblcks, nthrds>>>((real *) a,
					      (real *) b,
					      *c1, *n);

  }

  /**
   * Fortran wrapper for invcol2
   * Vector division \f$ a = a / b \f$
   */
  void cuda_invcol2(void *a, void *b, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    invcol2_kernel<real><<<nblcks, nthrds>>>((real *) a,
					       (real *) b, *n);
  }
  
  /**
   * Fortran wrapper for col2
   * Vector multiplication with 2 vectors \f$ a = a \cdot b \f$
   */
  void cuda_col2(void *a, void *b, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    col2_kernel<real><<<nblcks, nthrds>>>((real *) a, 
					    (real *) b, *n);
  }
  
  /**
   * Fortran wrapper for col3
   * Vector multiplication with 3 vectors \f$ a = b \cdot c \f$
   */
  void cuda_col3(void *a, void *b, void *c, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    col3_kernel<real><<<nblcks, nthrds>>>((real *) a, (real *) b,
					    (real *) c, *n);
  }
  

  /**
   * Fortran wrapper for sub3
   * Vector subtraction \f$ a = b - c \f$
   */
  void cuda_sub3(void *a, void *b, void *c, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    sub3_kernel<real><<<nblcks, nthrds>>>((real *) a, (real *) b, 
					    (real *) c, *n);
  }

  /**
   * Fortran wrapper for addcol3
   * \f$ a = a + b * c \f$
   */
  void cuda_addcol3(void *a, void *b, void *c, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);

    addcol3_kernel<real><<<nblcks, nthrds>>>((real *) a, (real *) b,
					       (real *) c, *n);
  }

  /**
   * Fortran wrapper glsc3
   * Weighted inner product \f$ a^T b c \f$
   */
  real cuda_glsc3(void *a, void *b, void *c, int *n) {
	
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);
    const int nb = ((*n) + 1024 - 1)/ 1024;
    
    real * buf = (real *) malloc(nb * sizeof(real));
    real * buf_d;

    hipMalloc(&buf_d, nb*sizeof(real));
     
    glsc3_kernel<real><<<nblcks, nthrds>>>((real *) a, (real *) b,
					     (real *) c, buf_d, *n);

    hipMemcpy(buf, buf_d, nb * sizeof(real), hipMemcpyDeviceToHost);

    real res = 0.0;
    for (int i = 0; i < nb; i++) {
      res += buf[i];
    }

    free(buf);
    hipFree(buf_d);

    return res;
  }

  /**
   * Fortran wrapper glsc3
   * Weighted inner product \f$ a^T b c \f$
   */
  real cuda_glsc2(void *a, void *b, int *n) {
	
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);
    const int nb = ((*n) + 1024 - 1)/ 1024;
    
    real * buf = (real *) malloc(nb * sizeof(real));
    real * buf_d;

    hipMalloc(&buf_d, nb*sizeof(real));
     
    glsc2_kernel<real><<<nblcks, nthrds>>>((real *) a, (real *) b,
					      buf_d, *n);

    hipMemcpy(buf, buf_d, nb * sizeof(real), hipMemcpyDeviceToHost);

    real res = 0.0;
    for (int i = 0; i < nb; i++) {
      res += buf[i];
    }

    free(buf);
    hipFree(buf_d);

    return res;
  }

}
