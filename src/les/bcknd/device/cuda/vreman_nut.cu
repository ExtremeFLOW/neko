/*
 Copyright (c) 2024, The Neko Authors
 All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions
 are met:

   * Redistributions of source code must retain the above copyright
     notice, this list of conditions and the following disclaimer.

   * Redistributions in binary form must reproduce the above
     copyright notice, this list of conditions and the following
     disclaimer in the documentation and/or other materials provided
     with the distribution.

   * Neither the name of the authors nor the names of its
     contributors may be used to endorse or promote products derived
     from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 POSSIBILITY OF SUCH DAMAGE.
*/

#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <device/device_config.h>
#include <device/cuda/check.h>
#include "vreman_nut_kernel.h"

extern "C" {
  #include <common/neko_log.h>
}

extern "C" {
  void cuda_vreman_nut_compute(void *a11, void *a12, void *a13,
                             void *a21, void *a22, void *a23,
                             void *a31, void *a32, void *a33, 
                             void *delta, void *nut, void *mult, 
                             real *c, real *eps, int * n){
    
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n)+1024 - 1)/ 1024, 1, 1);
    const hipStream_t stream = (hipStream_t) glb_cmd_queue;

    vreman_nut_compute<real>
    <<<nblcks, nthrds, 0, stream>>>((real *) a11, (real *) a12, (real *) a13,
                                    (real *) a21, (real *) a22, (real *) a23,
                                    (real *) a31, (real *) a32, (real *) a33,
                                    (real *) delta, (real *) nut, (real *) mult, 
                                    *c, * eps, * n);
    CUDA_CHECK(hipGetLastError());
  }
}