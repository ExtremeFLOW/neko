/*
 Copyright (c) 2022, The Neko Authors
 All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions
 are met:

   * Redistributions of source code must retain the above copyright
     notice, this list of conditions and the following disclaimer.

   * Redistributions in binary form must reproduce the above
     copyright notice, this list of conditions and the following
     disclaimer in the documentation and/or other materials provided
     with the distribution.

   * Neither the name of the authors nor the names of its
     contributors may be used to endorse or promote products derived
     from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 POSSIBILITY OF SUCH DAMAGE.
*/

#include <device/device_config.h>
#include <device/cuda/check.h>

#include "scalar_residual_update_kernel.h"

extern "C" {

  void scalar_residual_update_cuda(void *s_res, void *f_s, int *n) {

    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*n) + 1024 - 1) / 1024, 1, 1);

    scalar_residual_update_kernel<real>
      <<<nblcks, nthrds>>>((real *) s_res,
                           (real *) f_s, *n);
    CUDA_CHECK(hipGetLastError());
  }
    
}

