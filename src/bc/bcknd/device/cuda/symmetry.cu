#include "hip/hip_runtime.h"
/*
 Copyright (c) 2021-2022, The Neko Authors
 All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions
 are met:

   * Redistributions of source code must retain the above copyright
     notice, this list of conditions and the following disclaimer.

   * Redistributions in binary form must reproduce the above
     copyright notice, this list of conditions and the following
     disclaimer in the documentation and/or other materials provided
     with the distribution.

   * Neither the name of the authors nor the names of its
     contributors may be used to endorse or promote products derived
     from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 POSSIBILITY OF SUCH DAMAGE.
*/

#include <algorithm>
#include <device/device_config.h>
#include <device/cuda/check.h>

#include "symmetry_kernel.h"


extern "C" {

  /** 
   * Fortran wrapper for device symmetry apply vector
   */
  void cuda_symmetry_apply_vector(void *xmsk, void *ymsk, void *zmsk,
                                 void *x, void *y, void *z,
                                 int *m, int *n, int *l) {

    const int max_len = std::max(std::max(*m, *n), *l);
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((max_len) + 1024 - 1)/ 1024, 1, 1);

    symmetry_apply_vector_kernel<real>
      <<<nblcks, nthrds>>>((int *) xmsk, (int *) ymsk, (int *) zmsk,
                           (real *) x, (real *) y, (real *) z, *m, *n, *l);
    CUDA_CHECK(hipGetLastError());
  }
 
}
