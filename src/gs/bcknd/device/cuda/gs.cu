/*
 Copyright (c) 2021, The Neko Authors
 All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions
 are met:

   * Redistributions of source code must retain the above copyright
     notice, this list of conditions and the following disclaimer.

   * Redistributions in binary form must reproduce the above
     copyright notice, this list of conditions and the following
     disclaimer in the documentation and/or other materials provided
     with the distribution.

   * Neither the name of the authors nor the names of its
     contributors may be used to endorse or promote products derived
     from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 POSSIBILITY OF SUCH DAMAGE.
*/

#include <climits>
#include <cstdio>
#include <device/device_config.h>
#include <device/cuda/check.h>
#include "gs_kernels.h"

#define GS_OP_ADD  1
#define GS_OP_MUL  2
#define GS_OP_MIN  3
#define GS_OP_MAX  4

extern "C" {

  /** 
   * Fortran wrapper for device gather kernels
   */
  void cuda_gather_kernel(void *v, int *m, int *o, void *dg,
                          void *u, int *n, void *gd, int *nb,
                          void *b, void *bo, int *op) {

    if ((*m) == 0) return;
    
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*m)+ 1024 - 1)/ 1024, 1, 1);

    switch (*op) {
    case GS_OP_ADD:
      gather_kernel_add<real>
        <<<nblcks, nthrds>>>((real *) v, *m, *o, (int *) dg,
                             (real *) u, *n, (int *) gd,
                             *nb, (int *) b, (int *) bo);
      CUDA_CHECK(hipGetLastError());
      break;
    case GS_OP_MUL:
      gather_kernel_mul<real>
        <<<nblcks, nthrds>>>((real *) v, *m, *o, (int *) dg,
                             (real *) u, *n, (int *) gd,
                             *nb, (int *) b, (int *) bo);
      CUDA_CHECK(hipGetLastError());
      break;
    case GS_OP_MIN:
      gather_kernel_min<real>
        <<<nblcks, nthrds>>>((real *) v, *m, *o, (int *) dg,
                             (real *) u, *n, (int *) gd,
                             *nb, (int *) b, (int *) bo);
      CUDA_CHECK(hipGetLastError());
      break;
    case GS_OP_MAX:
      gather_kernel_max<real>
        <<<nblcks, nthrds>>>((real *) v, *m, *o, (int *) dg,
                             (real *) u, *n, (int *) gd,
                             *nb, (int *) b, (int *) bo);
      CUDA_CHECK(hipGetLastError());
      break;
    }
  }

  /**
   * Fortran wrapper for device scatter kernel
   */
  void cuda_scatter_kernel(void *v, int *m, void *dg,
                           void *u, int *n, void *gd,
                           int *nb, void *b, void *bo) {

    if ((*m) == 0) return;
        
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*m)+1024 - 1)/ 1024, 1, 1);

    scatter_kernel<real>
      <<<nblcks, nthrds>>>((real *) v, *m, (int *) dg,
                           (real *) u, *n, (int *) gd,
                           *nb, (int *) b, (int *) bo);
    CUDA_CHECK(hipGetLastError());
  }

  /**
   * Pack send buffer on device
   */
  void cuda_gs_pack(void *u_d, void *buf_d, void *dof_d,
                    int offset, int n, hipStream_t stream) {

    const int nthrds = 1024;
    const int nblcks = (n + nthrds - 1) / nthrds;

    if (stream == NULL) {
      gs_pack_kernel<real>
        <<<nblcks, nthrds>>>((real *) u_d, (real *) buf_d + offset,
                             (int *) dof_d + offset, n);
    }
    else {
      gs_pack_kernel<real>
        <<<nblcks, nthrds, 0, stream>>>((real *) u_d, (real *) buf_d + offset,
                                        (int *) dof_d + offset, n);
    }
      
    CUDA_CHECK(hipGetLastError());
  }

  /**
   * Unpack receive buffer on device
   */
  void cuda_gs_unpack(real *u_d, int op, real *buf_d, int *dof_d,
                      int offset, int n, hipStream_t stream) {

    const int nthrds = 1024;
    const int nblcks = (n + nthrds - 1) / nthrds;

    switch (op) {
    case GS_OP_ADD:
      if (stream == NULL) {
        gs_unpack_add_kernel<real>
          <<<nblcks, nthrds>>>(u_d, buf_d + offset, dof_d + offset, n);
      }
      else {
        gs_unpack_add_kernel<real>
          <<<nblcks, nthrds, 0, stream>>>(u_d, buf_d + offset,
                                          dof_d + offset, n);
      }
      break;
    default:
      printf("%s: unknown gs op %d\n", __FILE__, op);
      abort();
    }

    CUDA_CHECK(hipGetLastError());
  }
}
