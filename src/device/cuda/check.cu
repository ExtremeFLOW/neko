#include <stdio.h>
#include <device/cuda/check.h>

/**
 * Check a CUDA return code
 */
void cuda_check(const char *fname, const int line, const hipError_t err)
{
  if (err != hipSuccess) {
    fprintf(stderr, "%s in %s:%d \n", hipGetErrorString(err), fname, line);
    exit(1);
  }						  
}
